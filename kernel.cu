#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__
void work(int n, long long *x, long long *y){
    //printf("IN KERNEL\n");
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i+=stride){
        long long sum = 0;
        for (long long j = 0; j <= x[i]; j++){
            if (x[i] % j == 0){
                //is a divisor
                sum += j * j;
            }
        }
        double sr = round(sqrt((double) sum));
        if((double)sum == sr*sr){
            y[i] = 1;
        } else {
            y[i] = 0;
        }
    }
}

void run(){
    //int N = 64000001;
    int N = 100000;
    long long *x, *y;
    hipMallocManaged(&x, N*sizeof(long long));
    hipMallocManaged(&y, N*sizeof(long long));
    for (int i = 0; i < N; i++){
        x[i] = i;
        y[i] = 0;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    work<<<numBlocks,blockSize>>>(N, x, y);
    hipDeviceSynchronize();
    long long s = 0;
    for (int i = 0; i < N; i++){
        if (y[i]){
            std::cout<<x[i]<<'\n';
            s += x[i];
        }
    }
    std::cout<<"Sum: "<<s<<'\n';
    hipFree(x);
    hipFree(y);
}