#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__
void work(int n, int *y, long long *squares){
    //printf("IN KERNEL\n");
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int count = 0;
    for(int i = index; i < n; i+=stride){
        //count++;
        //if(count % 1000 == 0){printf("%d", count);}
        long long sum = 0;        
        long long j = 1;
        while (j * j <= i){
            if (i % j == 0){
                if  (!squares[j]){squares[j]=j*j;}
                sum += squares[j];
                if (j * j != i){
                    if  (!squares[(i/j)]){squares[(i/j)]=(i/j)*(i/j);}
                    sum += (squares[(i/j)]);
                }
            }
            j++;
        }
        
        double sr = round(sqrt((double) sum));
        if((double)sum == sr*sr){
            y[i] = 1;
        } else {
            y[i] = 0;
        }
    }
}

void run(){
    //int N = 64000001;
    int N = 5000000;
    //int *x, *y;
    int *y;
    long long *squares;
    //hipMallocManaged(&x, N*sizeof(int));
    hipMallocManaged(&y, N*sizeof(int));
    hipMallocManaged(&squares, N*sizeof(long long));
    for (int i = 0; i < N; i++){
        //x[i] = i;
        y[i] = 0;
        squares[i] = 0;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    work<<<numBlocks,blockSize>>>(N, y, squares);
    hipDeviceSynchronize();
    long long s = 0;
    for (int i = 0; i < N; i++){
        if (y[i]){
            std::cout<<i<<'\n';
            s += i;
        }
    }
    std::cout<<"Sum: "<<s<<'\n';
    //hipFree(x);
    hipFree(y);
    hipFree(squares);
}